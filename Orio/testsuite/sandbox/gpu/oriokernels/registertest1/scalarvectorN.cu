#include "hip/hip_runtime.h"
//scalarvectorN.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
//#include <omp.h>


#define SEED 1

double getclock();
void checkCUDAError(const char *);
__global__ void sVPN(float *vector, float* scalar, int* vlength, int *N);
__global__ void sVPNo(float *vector, float* scalar, int* vlength, int *N);






int main(int argc, char** argv){

        printf("\n\nARGC value: %d\n",argc);
        int N = atoi(argv[1]);
	int len = atoi(argv[2]);
	int tpb = atoi(argv[3]);
	if(argc<5){
	  srand(SEED);
	}else{
	  srand(atoi(argv[4]));
	}

	
	int i;


//	Get device information
	int count;
	hipGetDeviceCount(&count);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	printf("Compute capability: %d.%d\n",prop.major,prop.minor);
	printf("Number of GPUs: %d\n",count);
	printf("Multiprocessor count: %d\n",prop.multiProcessorCount);
	printf("Clock rate: %luKhz\n",prop.clockRate/1000);
	printf("Total Global Memory: %luMB\n",(unsigned int)prop.totalGlobalMem/1000000);
	printf("Total Constant Memory: %d\n",prop.totalConstMem);
	printf("Shared memory per block: %d\n",prop.sharedMemPerBlock);
	printf("1-D Texture Max size: %d\n",prop.maxTexture1D);
	printf("Number of registers per block: %d\n",prop.regsPerBlock);
	printf("Can I map host memory: %d\n",prop.canMapHostMemory);
	printf("Max number of threads per block: %d\n",prop.maxThreadsPerBlock);
	printf("Max number of blocks in a grid [0]: %d\n",prop.maxGridSize[0]);
	printf("Max number of blocks in a grid [1]: %d\n",prop.maxGridSize[1]);
	printf("Max number of blocks in a grid [2]: %d\n",prop.maxGridSize[2]);
	printf("Max Texture dimensions 2D: %lu\n",prop.maxTexture2D[2]);
	printf("Concurrent Kernels: %d\n",prop.concurrentKernels);
	printf("Threads in a warp: %d\n",prop.warpSize);

//	some general cpu info
	printf("size of float (cpu): %d\n",sizeof(float));
	printf("size of unsigned int (cpu): %d\n",sizeof(unsigned int));
	printf("size of unsigned long (cpu): %d\n",sizeof(unsigned long));


	float scalar=100.00/(rand()%100+1);

	float *vector;
	vector=(float*)malloc(len*sizeof(float));

	for(i=0;i<len;i++){
	    vector[i]=100.00/(rand()%100+1);
	}

	hipError_t cudastatus0,cudastatus1;

	float* devVec;
	cudastatus0=hipMalloc((void**)&devVec,len*sizeof(float));
	cudastatus1=hipMemcpy(devVec,vector,len*sizeof(float),hipMemcpyHostToDevice);
	if(cudastatus0!=hipSuccess|cudastatus1!=hipSuccess){
	  printf("Error in devVec memory allocation:\nstatus0: %s, status1: %s\n",
  			hipGetErrorString(cudastatus0),
			hipGetErrorString(cudastatus1));
	  if(devVec) hipFree(devVec);
	  if(vector) free(vector);
          exit(1);
	}

	float* devScal;
	cudastatus0=hipMalloc((void**)&devScal,sizeof(float));
	cudastatus1=hipMemcpy(devScal,&scalar,sizeof(float),hipMemcpyHostToDevice);
	if(cudastatus0!=hipSuccess|cudastatus1!=hipSuccess){
	  printf("Error in devScal memory allocation:\nstatus0: %s, status1: %s\n",
  			hipGetErrorString(cudastatus0),
			hipGetErrorString(cudastatus1));
	  if(devVec) hipFree(devVec);
	  if(devScal) hipFree(devScal);
	  if(vector) free(vector);
          exit(1);
	}

	int* devLen;
	cudastatus0=hipMalloc((void**)&devLen,sizeof(int));
	cudastatus1=hipMemcpy(devLen,&len,sizeof(int),hipMemcpyHostToDevice);
	if(cudastatus0!=hipSuccess|cudastatus1!=hipSuccess){
	  printf("Error in devLen memory allocation:\nstatus0: %s, status1: %s\n",
  			hipGetErrorString(cudastatus0),
			hipGetErrorString(cudastatus1));
	  if(devVec) hipFree(devVec);
	  if(devScal) hipFree(devScal);
	  if(vector) free(vector);
          exit(1);
	}

	int* devN;
	cudastatus0=hipMalloc((void**)&devN,sizeof(int));
	cudastatus1=hipMemcpy(devN,&N,sizeof(int),hipMemcpyHostToDevice);
	if(cudastatus0!=hipSuccess|cudastatus1!=hipSuccess){
	  printf("Error in devN memory allocation:\nstatus0: %s, status1: %s\n",
  			hipGetErrorString(cudastatus0),
			hipGetErrorString(cudastatus1));
	  if(devVec) hipFree(devVec);
	  if(devScal) hipFree(devScal);
	  if(devLen) hipFree(devLen);
	  if(devN) hipFree(devN);
	  if(vector) free(vector);
          exit(1);
	}


	printf("\n\nVector size: %dK Iterations: %dK Memory: %dKB\n",len/1000,N/1000,len*sizeof(float)/1024);

	int blocks=ceil((float)len/(float)tpb);
	int threads=tpb;
	printf("Number of blocks: %d, threads per block: %d\n",blocks,threads);

	cudastatus0=hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	if(cudastatus0!=hipSuccess){
	  printf("Error in setting L1 cache level to prefer Shared, status0: %s\n",
	  		hipGetErrorString(cudastatus0));
	}

	float elapsedtime1;
	hipEvent_t start1,stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1,0);//begin recording kernel
  	sVPN<<<blocks,threads>>>(devVec,devScal,devLen,devN);
	hipEventRecord(stop1,0);
	hipEventSynchronize(stop1); // event barrier
	hipEventElapsedTime(&elapsedtime1,start1,stop1);
        hipEventDestroy(start1);
	hipEventDestroy(stop1);


	float elapsedtime2;
	hipEvent_t start2,stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2,0);//begin recording kernel
  	sVPNo<<<blocks,threads>>>(devVec,devScal,devLen,devN);
	hipEventRecord(stop2,0);
	hipEventSynchronize(stop2); // event barrier
	hipEventElapsedTime(&elapsedtime2,start2,stop2);
        hipEventDestroy(start2);
	hipEventDestroy(stop2);


	cudastatus0=hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	if(cudastatus0!=hipSuccess){
	  printf("Error in setting L1 cache level to Cache preferred, status0: %s\n",
	  		hipGetErrorString(cudastatus0));
	}

	float elapsedtime3;
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3,0);//begin recording kernel
  	sVPN<<<blocks,threads>>>(devVec,devScal,devLen,devN);
	hipEventRecord(stop3,0);
	hipEventSynchronize(stop3); // event barrier
	hipEventElapsedTime(&elapsedtime3,start3,stop3);
        hipEventDestroy(start3);
	hipEventDestroy(stop3);


	float elapsedtime4;
	hipEvent_t start4,stop4;
	hipEventCreate(&start4);
	hipEventCreate(&stop4);
	hipEventRecord(start4,0);//begin recording kernel
  	sVPNo<<<blocks,threads>>>(devVec,devScal,devLen,devN);
	hipEventRecord(stop4,0);
	hipEventSynchronize(stop4); // event barrier
	hipEventElapsedTime(&elapsedtime4,start4,stop4);
        hipEventDestroy(start4);
	hipEventDestroy(stop4);



	printf("\nunoptimized kernel 16KB L1: %lf msec.\n",elapsedtime1);
	printf("optimized kernel 16KB L1: %lf msec.\n",elapsedtime2);
	printf("unoptimized kernel 48KB L1: %lf msec.\n",elapsedtime3);
	printf("optimized kernel 48KB L1: %lf msec.\n\n\n",elapsedtime4);

	if(devScal) hipFree(devScal);
	if(devLen) hipFree(devLen);
	if(devN) hipFree(devN);
	if(devVec) hipFree(devVec);
	if(vector) free(vector);
	checkCUDAError("cuda free operations");
	printf("Exiting...\n\n");
	exit(0);
}




// .................................................................
// Unoptimized code - kernel multiplies a vector by a scalar N times
// .................................................................
//
__global__ void sVPN(float *vector, float* scalar, int* vlength, int *N){

  int tid = blockDim.x*blockIdx.x+threadIdx.x;

  unsigned int i;// using unsigned type for loop counter will decrease performance
  
  for(i=0;i<*N;i++){// per loop global access of N if not in register
  	if(tid<*vlength)vector[tid]*=*scalar;//per loop global access of scalar and vlength 
  }
  
  //3N global memory accesses per thread or 4N if N is not in register
}




// .................................................................
// Optimized code - kernel multiplies a vector by a scalar N times
// 	 replacing global memory access with register(local) variables
// .................................................................
//
__global__ void sVPNo(float *vector, float* scalar, int* vlength, int *N){

  int tid = blockDim.x*blockIdx.x+threadIdx.x;

  //local variables added
  float local_scalar = *scalar;
  float local_N = *N;
  int local_vlength=*vlength;
  int local_vector= vector[tid];
  
  int i;// transform to int
  
  for(i=0;i< local_N;i++){
  	if(tid<local_vlength)local_vector*=local_scalar; 
  }
  vector[tid]=local_vector;
  //5 global memory accesses per thread
}



void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 



double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}
