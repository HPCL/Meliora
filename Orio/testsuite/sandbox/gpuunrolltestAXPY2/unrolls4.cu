#include "hip/hip_runtime.h"
#include "book.h"

#define UF 4

__global__ void orcu_kernel6(int n, int orcu_var3, double a1, double a2, double* y, double* x1, double* x2) {
  int tid=UF*(blockIdx.x*blockDim.x+threadIdx.x)+orcu_var3;
  if (tid<=n-UF) {
    {
      y[tid]=y[tid]+a1*x1[tid]+a2*x2[tid];
      int index = tid+1;
      y[index]=y[index]+a1*x1[index]+a2*x2[index];
      index = tid+2;
      y[index]=y[index]+a1*x1[index]+a2*x2[index];
      index = tid+3;
      y[index]=y[index]+a1*x1[index]+a2*x2[index];
    }
  }
}
//__global__ void orcu_kernel11(int n, int orcu_var8, double a1, double* y, double* x1) {
  //int tid=blockIdx.x*blockDim.x+threadIdx.x+orcu_var8;
  //if (tid<=n-1) {
    //y[tid]=y[tid]+a1*x1[tid];
  //}
//}


void axpy1(int n, double *y, double a1, double a2, double *x1, double *x2)
{
register int i;


/*@ begin Loop(
  transform Composite(
    cuda = (16,False, False, 1)
    ,scalarreplace = (False, 'int')
, unrolljam = (['i'], [2])
  )
   {
    for (i=0; i<=n-1; i++) {
    	y[i]=y[i]+a1*x1[i];
    }
    
   }


   
  
) @*/

hipEvent_t start, stop;
HANDLE_ERROR(hipEventCreate(&start));
HANDLE_ERROR(hipEventCreate(&stop));


{
  {
    int orio_lbound1=0;
    //{
      /*declare variables*/
      double *dev_y, *dev_x1, *dev_x2;
      int nthreads=TC;
      /*calculate device dimensions*/
      dim3 dimGrid, dimBlock;
      dimBlock.x=nthreads;
      dimGrid.x=(n+nthreads-1)/nthreads;
      dimGrid.x=(dimGrid.x+UF-1)/UF;
      printf("num of blocks: %d\n", dimGrid.x);
      /*allocate device memory*/
      int nbytes=n*sizeof(double);
      hipMalloc((void**)&dev_y,nbytes);
      hipMalloc((void**)&dev_x1,nbytes);
      hipMalloc((void**)&dev_x2,nbytes);
      /*copy data from host to device*/
      hipMemcpy(dev_y,y,nbytes,hipMemcpyHostToDevice);
      hipMemcpy(dev_x1,x1,nbytes,hipMemcpyHostToDevice);
      hipMemcpy(dev_x2,x2,nbytes,hipMemcpyHostToDevice);
      /*invoke device kernel*/
      int orcu_var3=orio_lbound1;

      HANDLE_ERROR(hipEventRecord(start, 0));
      orcu_kernel6<<<dimGrid,dimBlock>>>(n,orcu_var3,a1,a2,dev_y,dev_x1, dev_x2);
      HANDLE_ERROR(hipEventRecord(stop, 0));
      /*copy data from device to host*/
      hipMemcpy(y,dev_y,nbytes,hipMemcpyDeviceToHost);
      /*free allocated memory*/
      hipFree(dev_y);
      hipFree(dev_x1);
      hipFree(dev_x2);
    //}
    //int orio_lbound2=n-((n-(0))%2);
    {
      /*declare variables*/
      //double *dev_y, *dev_x1;
      //int nthreads=TC;
      /*calculate device dimensions*/
      //dim3 dimGrid, dimBlock;
      //dimBlock.x=nthreads;
      //dimGrid.x=(n+nthreads-1)/nthreads;
      /*allocate device memory*/
      //int nbytes=n*sizeof(double);
      //hipMalloc((void**)&dev_y,nbytes);
      //hipMalloc((void**)&dev_x1,nbytes);
      /*copy data from host to device*/
      //hipMemcpy(dev_y,y,nbytes,hipMemcpyHostToDevice);
      //hipMemcpy(dev_x1,x1,nbytes,hipMemcpyHostToDevice);
      /*invoke device kernel*/
      //int orcu_var8=orio_lbound2;
      //orcu_kernel11<<<dimGrid,dimBlock>>>(n,orcu_var8,a1,dev_y,dev_x1);
      /*copy data from device to host*/
      //hipMemcpy(y,dev_y,nbytes,hipMemcpyDeviceToHost);
      /*free allocated memory*/
      //hipFree(dev_y);
      //hipFree(dev_x1);
    }
  }
}
/*@ end @*/
HANDLE_ERROR(hipEventSynchronize(stop));
float passedTime;
HANDLE_ERROR(hipEventElapsedTime(&passedTime, start, stop));
HANDLE_ERROR(hipEventDestroy(start));
HANDLE_ERROR(hipEventDestroy(stop));
printf("timePassed: %f ms\n", passedTime);
}

int main(){
	double* y = (double*) malloc(sizeof(double)*NN);
	double* x1 = (double*) malloc(sizeof(double)*NN);
	double* x2 = (double*) malloc(sizeof(double)*NN);
	double a1 = AA;
	double a2 = AA2;
	int i;
        
	for(i=0; i<NN; i++){
		y[i] = i;
		x1[i] = i;
		x2[i] = i;
	}
	axpy1(NN, y, a1, a2, x1, x2);
	for(i=0; i<13; i++)
		printf("%f\n", y[i]);
        for(i=NN-9; i<NN; i++)
                printf("%f\n", y[i]);

	return 0;
}
