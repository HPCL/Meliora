#include "hip/hip_runtime.h"
#include "book.h"

#define UF 4

__global__ void orcu_kernel6(int n, int orcu_var3, double a1, double* y, double* x1) {
  int tid=blockIdx.x*blockDim.x+threadIdx.x+orcu_var3;
  int k=(n/UF);
  if (tid<=orcu_var3+k-1) {
    {
      y[tid]=y[tid]+a1*x1[tid];
      int index = tid+k;
      y[index]=y[index]+a1*x1[index];
      index = tid+2*k;
      y[index]=y[index]+a1*x1[index];
      index = tid+3*k;
      y[index]=y[index]+a1*x1[index];
    }
  }
}
//__global__ void orcu_kernel11(int n, int orcu_var8, double a1, double* y, double* x1) {
  //int tid=blockIdx.x*blockDim.x+threadIdx.x+orcu_var8;
  //if (tid<=n-1) {
    //y[tid]=y[tid]+a1*x1[tid];
  //}
//}


void axpy1(int n, double *y, double a1, double *x1)
{
register int i;


/*@ begin Loop(
  transform Composite(
    cuda = (16,False, False, 1)
    ,scalarreplace = (False, 'int')
, unrolljam = (['i'], [2])
  )
   {
    for (i=0; i<=n-1; i++) {
    	y[i]=y[i]+a1*x1[i];
    }
    
   }


   
  
) @*/

hipEvent_t start, stop;
HANDLE_ERROR(hipEventCreate(&start));
HANDLE_ERROR(hipEventCreate(&stop));

{
  {
    int orio_lbound1=0;
    //{
      /*declare variables*/
      double *dev_y, *dev_x1;
      int nthreads=TC;
      /*calculate device dimensions*/
      dim3 dimGrid, dimBlock;
      dimBlock.x=nthreads;
      dimGrid.x=(n+nthreads-1)/nthreads;
      dimGrid.x=(dimGrid.x+UF-1)/UF;
      printf("num of blocks: %d\n", dimGrid.x);
      /*allocate device memory*/
      int nbytes=n*sizeof(double);
      hipMalloc((void**)&dev_y,nbytes);
      hipMalloc((void**)&dev_x1,nbytes);
      /*copy data from host to device*/
      hipMemcpy(dev_y,y,nbytes,hipMemcpyHostToDevice);
      hipMemcpy(dev_x1,x1,nbytes,hipMemcpyHostToDevice);
      /*invoke device kernel*/
      int orcu_var3=orio_lbound1;
      HANDLE_ERROR(hipEventRecord(start, 0));
      orcu_kernel6<<<dimGrid,dimBlock>>>(n,orcu_var3,a1,dev_y,dev_x1);
      HANDLE_ERROR(hipEventRecord(stop, 0));
      HANDLE_ERROR(hipEventSynchronize(stop));
      /*copy data from device to host*/
      hipMemcpy(y,dev_y,nbytes,hipMemcpyDeviceToHost);
      /*free allocated memory*/
      hipFree(dev_y);
      hipFree(dev_x1);
    //}
    //int orio_lbound2=n-((n-(0))%UF);
    {
      /*declare variables*/
      //double *dev_y, *dev_x1;
      //int nthreads=TC;
      /*calculate device dimensions*/
      //dim3 dimGrid, dimBlock;
      //dimBlock.x=nthreads;
      //dimGrid.x=(n+nthreads-1)/nthreads;
      /*allocate device memory*/
      //int nbytes=n*sizeof(double);
      //hipMalloc((void**)&dev_y,nbytes);
      //hipMalloc((void**)&dev_x1,nbytes);
      /*copy data from host to device*/
      //hipMemcpy(dev_y,y,nbytes,hipMemcpyHostToDevice);
      //hipMemcpy(dev_x1,x1,nbytes,hipMemcpyHostToDevice);
      /*invoke device kernel*/
      //int orcu_var8=orio_lbound2;
      //orcu_kernel11<<<dimGrid,dimBlock>>>(n,orcu_var8,a1,dev_y,dev_x1);
      /*copy data from device to host*/
      //hipMemcpy(y,dev_y,nbytes,hipMemcpyDeviceToHost);
      /*free allocated memory*/
      //hipFree(dev_y);
      //hipFree(dev_x1);
    }
  }
}
/*@ end @*/


float passedTime;
HANDLE_ERROR(hipEventElapsedTime(&passedTime, start, stop));
HANDLE_ERROR(hipEventDestroy(start));
HANDLE_ERROR(hipEventDestroy(stop));
printf("timePassed: %f ms\n", passedTime);
}

int main(){
	double* y = (double*) malloc(sizeof(double)*NN);
	double* x1 = (double*) malloc(sizeof(double)*NN);
	double a1 = AA;
	int i;
        
	for(i=0; i<NN; i++){
		y[i] = i;
		x1[i] = i;
	}
	axpy1(NN, y, a1, x1);
	for(i=0; i<13; i++)
		printf("%f\n", y[i]);
        for(i=NN-9; i<NN; i++)
                printf("%f\n", y[i]);

	return 0;
}
