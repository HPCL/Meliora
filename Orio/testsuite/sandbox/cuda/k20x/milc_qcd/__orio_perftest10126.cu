#include "hip/hip_runtime.h"
/*
CFLAGS:-O3
UIF:5
TC:448
PL:48
BC:70
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define SITES 2
double *A;
double *x;
double *y;
void malloc_arrays() {
  int i1;
  A = (double*) malloc((18 *SITES) * sizeof(double));
  x = (double*) malloc((6 *SITES) * sizeof(double));
  y = (double*) malloc((6 *SITES) * sizeof(double));
}

void init_input_vars() {
  int i1;
  for (i1=0; i1<18 *SITES; i1++)
   A[i1] = (i1) % 5 + 1;
  for (i1=0; i1<6 *SITES; i1++)
   x[i1] = (i1) % 5 + 1;
  for (i1=0; i1<6 *SITES; i1++)
   y[i1] = 0;
}



__global__ void orcu_kernel30377(const int sites_on_node, double* A, double* y, double* x) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  double ci, ai, bi, ar, br, cr;
  int j, k;
  for (int i=tid; i<=sites_on_node-1; i+=gsize) {
    {
      #pragma unroll 5
      for (j=0; j<=5; j=j+2) {
        cr=ci=0.0;
        for (k=0; k<=5; k=k+2) {
          ar=A[18*i+3*j+k];
          ai=A[18*i+3*j+k+1];
          br=x[6*i+k];
          bi=x[6*i+k+1];
          cr=cr+ar*br-ai*bi;
          ci=ci+ar*bi+ai*br;
        }
        y[6*i+j]=cr;
        y[6*i+j+1]=ci;
      }
    }
  }
}


int main(int argc, char *argv[]) {
  malloc_arrays();
  init_input_vars();

  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  float orcu_elapsed=0.0, orcu_transfer=0.0;
  hipEvent_t tstart, tstop, start, stop;
  hipEventCreate(&tstart); hipEventCreate(&tstop);
  hipEventCreate(&start);  hipEventCreate(&stop);
  for (int orio_i=0; orio_i<ORIO_REPS; orio_i++) {
    

  int sites_on_node=SITES;

  /*@ begin Loop(transform CUDA(threadCount=TC, blockCount=BC, preferL1Size=PL, unrollInner=UIF)

  for(i=0; i<=sites_on_node-1; i++) {
    for(j=0; j<=5; j+=2) {
      cr = ci = 0.0;
      for(k=0; k<=5; k+=2) {
        ar=A[18*i+3*j+k];
        ai=A[18*i+3*j+k+1];
        br=x[6*i+k];
        bi=x[6*i+k+1];
        cr += ar*br - ai*bi;
        ci += ar*bi + ai*br;
      }
      y[6*i+j]  =cr;
      y[6*i+j+1]=ci;
    }
  }

  ) @*/
  {
    hipDeviceSynchronize();
    /*declare variables*/
    double *dev_A, *dev_y, *dev_x;
    int nthreads=448;
    /*calculate device dimensions*/
    dim3 dimGrid, dimBlock;
    dimBlock.x=nthreads;
    dimGrid.x=70;
    /*allocate device memory*/
    hipMalloc(&dev_A,18 *SITES*sizeof(double));
    hipMalloc(&dev_x,6 *SITES*sizeof(double));
    hipMalloc(&dev_y,6 *SITES*sizeof(double));
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    /*copy data from host to device*/
    hipEventRecord(tstart,0);
    hipMemcpy(dev_A,A,18 *SITES*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_x,x,6 *SITES*sizeof(double),hipMemcpyHostToDevice);
    hipEventRecord(tstop,0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&orcu_transfer,tstart,tstop);
    hipEventRecord(start,0);
    /*invoke device kernel*/
    orcu_kernel30377<<<dimGrid,dimBlock>>>(sites_on_node,dev_A,dev_y,dev_x);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&orcu_elapsed,start,stop);
    /*copy data from device to host*/
    hipMemcpy(y,dev_y,6 *SITES*sizeof(double),hipMemcpyDeviceToHost);
    hipDeviceSetCacheConfig(hipFuncCachePreferNone);
    /*free allocated memory*/
    hipFree(dev_A);
    hipFree(dev_y);
    hipFree(dev_x);
    hipError_t err=hipGetLastError();
    if (hipSuccess!=err) 
      printf("CUDA runtime error: %s@",hipGetErrorString(err));
  }
/*@ end @*/
  
    printf("{'[13, 4, 4, 1, 3]' : (%g,%g)}\n", orcu_elapsed, orcu_transfer);
  }
  hipEventDestroy(tstart); hipEventDestroy(tstop);
  hipEventDestroy(start);  hipEventDestroy(stop);
  
  
  return 0;
}
